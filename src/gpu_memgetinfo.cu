#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifndef GPU_MEMGETINFO
#define GPU_MEMGETINFO

void gpu_memgetinfo(int dev, unsigned int* freeMem, unsigned int *totalMem)
{
	float *f = NULL;
	hipMalloc((void**)&f, sizeof(float));
	unsigned int fm = 0;
	unsigned int mm = 0;
	hipMemGetInfo(&fm, &mm);
	*freeMem = fm;
	*totalMem = mm;
	hipFree(f);	
}

#endif